#include "hip/hip_runtime.h"
#pragma once
#include <precomp.h>

using namespace glm;
/////////////////////
/////BVH NODE //////
////////////////////

void BVHNode::CalculateBounds(std::vector<AABB> AABBs) {
	if (AABBs.size() <= 0) {
		return;
	}
	AABB aabb;
	aabb = AABBs[0];

	for (int i = 0; i < AABBs.size(); i++) {
		if (aabb.bounds[1].x < AABBs[i].bounds[1].x) {
			aabb.bounds[1].x = AABBs[i].bounds[1].x;
		}

		if (aabb.bounds[0].x > AABBs[i].bounds[0].x) {
			aabb.bounds[0].x = AABBs[i].bounds[0].x;
		}

		if (aabb.bounds[1].y < AABBs[i].bounds[1].y) {
			aabb.bounds[1].y = AABBs[i].bounds[1].y;
		}

		if (aabb.bounds[0].y > AABBs[i].bounds[0].y) {
			aabb.bounds[0].y = AABBs[i].bounds[0].y;
		}

		if (aabb.bounds[1].z < AABBs[i].bounds[1].z) {
			aabb.bounds[1].z = AABBs[i].bounds[1].z;
		}

		if (aabb.bounds[0].z > AABBs[i].bounds[0].z) {
			aabb.bounds[0].z = AABBs[i].bounds[0].z;
		}
	}

	aabb.CalculateCentre();
	this->aabb = aabb;

	//printf("%f %f %f | %f %f %f \n", aabb.bounds[0].x, aabb.bounds[0].y, aabb.bounds[0].z, aabb.bounds[1].x, aabb.bounds[1].y, aabb.bounds[1].z);
}

/////////////
////AABB/////
/////////////
void AABB::CalculateCentre() {
	this->centre_point = vec3((bounds[0].x + bounds[1].x) / 2.0f, (bounds[0].y + bounds[1].y) / 2.0f, (bounds[0].z + bounds[1].z) / 2.0f);
}

///////////////
/////BVH //////
//////////////

void BVH::ConstructBVH(std::vector<vec3> t_vertices, std::vector<int> t_indices, int tri_count) {

	std::vector<AABB> AABBs = std::vector<AABB>();
	//indices = new int[tri_count];

	for (int i = 0; i < tri_count; i++) {
		//indices[i] = i;

		vec3 v1 = t_vertices[t_indices[i * 3 + 1]];
		vec3 v2 = t_vertices[t_indices[i * 3 + 2]];
		vec3 v3 = t_vertices[t_indices[i * 3]];

		AABB bounds = AABB();
		bounds.bounds[1].x = max(v1.x, max(v2.x, v3.x));
		bounds.bounds[0].x = min(v1.x, min(v2.x, v3.x));
		bounds.bounds[1].y = max(v1.y, max(v2.y, v3.y));
		bounds.bounds[0].y = min(v1.y, min(v2.y, v3.y));
		bounds.bounds[1].z = max(v1.z, max(v2.z, v3.z));
		bounds.bounds[0].z = min(v1.z, min(v2.z, v3.z));
		bounds.CalculateCentre();
		bounds.index = i;
		AABBs.push_back(bounds);
	}

	//create the root node of the tree
	this->root_node = new BVHNode();
	this->root_node->CalculateBounds(AABBs);

	//subdivide to create the child nodes recursively
	Subdivide(root_node, AABBs, 1);
}

void BVH::Subdivide(BVHNode* node, std::vector<AABB> AABBs, bool side) {
	//base case, this is a leaf node if there are less than 3 objects inside
	if (AABBs.size() < 4) {
		node->is_leaf = true;
		for (int i = 0; i < AABBs.size(); i++) {
			node->leaf_triangles.push_back(AABBs[i].index);
		}
		return;
	}

	node->left = new BVHNode();
	node->right = new BVHNode();

	//calculate partition
	float split_position = -1;
	int split = Partition(node, AABBs, split_position);

	//if no split was made because child nodes do not improve cost then make this a leaf
	if (split == -1) {
		node->is_leaf = true;
		for (int i = 0; i < AABBs.size(); i++) {
			node->leaf_triangles.push_back(AABBs[i].index);
		}
		return;
	}
	
	//else subdivide
	std::vector<AABB> left_AABBs = std::vector<AABB>();
	std::vector<AABB> right_AABBs = std::vector<AABB>();

	float centre_value;
	if (split == 0) {
		centre_value = node->aabb.centre_point.x;
	}
	else if (split == 1) {
		centre_value = node->aabb.centre_point.y;
	}
	else {
		centre_value = node->aabb.centre_point.z;
	}

	for (int i = 0; i < AABBs.size(); i++) {
		float centre_check;
		if (split == 0) {
			centre_check = AABBs[i].centre_point.x;
		}
		else if (split == 1) {
			centre_check = AABBs[i].centre_point.y;
		}
		else {
			centre_check = AABBs[i].centre_point.z;
		}

		if (centre_check <= centre_value) {
			left_AABBs.push_back(AABBs[i]);
		}
		else {
			right_AABBs.push_back(AABBs[i]);
		}
	}


	node->left->CalculateBounds(left_AABBs);
	node->right->CalculateBounds(right_AABBs);

	if (left_AABBs.size() == 0 || right_AABBs.size() == 0) {
		node->is_leaf = true;
		for (int i = 0; i < AABBs.size(); i++) {
			node->leaf_triangles.push_back(AABBs[i].index);
		}
		return;
	}
	//recursive calls to partition child nodes
	Subdivide(node->left, left_AABBs, 1);
	Subdivide(node->right, right_AABBs, 0);

	node->is_leaf = false;

}

int BVH::Partition(BVHNode* node, std::vector<AABB> aabbs, float& split_position) {
	/*--FINDING SPLIT AXIS--*/
	//Using the Arbitrary Acyclic Algorithm from: http://graphicsinterface.org/wp-content/uploads/gi1989-22.pdf
	//spatial medians
	float xSpatialMedian = (node->aabb.bounds[0].x + node->aabb.bounds[1].x) / 2.0f;
	float ySpatialMedian = (node->aabb.bounds[0].y + node->aabb.bounds[1].y) / 2.0f;
	float zSpatialMedian = (node->aabb.bounds[0].z + node->aabb.bounds[1].z) / 2.0f;

	//object medians
	float xObjectMedian;
	float yObjectMedian;
	float zObjectMedian;

	std::vector<float> xVals = std::vector<float>();
	std::vector<float> yVals = std::vector<float>();
	std::vector<float> zVals = std::vector<float>();
	xVals.reserve(aabbs.size());
	yVals.reserve(aabbs.size());
	zVals.reserve(aabbs.size());

	for (int i = 0; i < aabbs.size(); i++) {
		xVals[i] = aabbs[i].centre_point.x;
		yVals[i] = aabbs[i].centre_point.y;
		zVals[i] = aabbs[i].centre_point.z;
	}

	std::sort(xVals.begin(), xVals.end());
	std::sort(yVals.begin(), yVals.end());
	std::sort(zVals.begin(), zVals.end());


	xObjectMedian = xVals[aabbs.size() / 2];
	yObjectMedian = yVals[aabbs.size() / 2];
	zObjectMedian = zVals[aabbs.size() / 2];

	//choose 9 equally spaced possibilities on each axis and check cost.
	float height = node->aabb.bounds[1].y - node->aabb.bounds[0].y;
	float width = node->aabb.bounds[1].x - node->aabb.bounds[0].x;
	float depth = node->aabb.bounds[1].z - node->aabb.bounds[0].z;

	float SA = 2 * (height * width) + 2 * (height * depth) + 2 * (width * depth);
	float parentCost = SA * aabbs.size();
	float splitCost = parentCost;
	char splitAxis = 'x';
	float splitPosition = xSpatialMedian;
	//x axis
	float min = fminf(xObjectMedian, xSpatialMedian);
	float max = fmaxf(xObjectMedian, xSpatialMedian);
	float step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float widthL = split - node->aabb.bounds[0].x;
		float widthR = node->aabb.bounds[1].x - split;

		float SALeft = 2 * (height * widthL) + 2 * (height * depth) + 2 * (widthL * depth);
		float SARight = 2 * (height * widthR) + 2 * (height * depth) + 2 * (widthR * depth);

		int n = 0;
		while (xVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)aabbs.size() - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'x';
			splitPosition = split;
		}
	}

	//y axis
	min = fminf(yObjectMedian, ySpatialMedian);
	max = fmaxf(yObjectMedian, ySpatialMedian);
	step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float heightL = split - node->aabb.bounds[0].y;
		float heightR = node->aabb.bounds[1].y - split;

		float SALeft = 2 * (heightL * width) + 2 * (heightL * depth) + 2 * (width * depth);
		float SARight = 2 * (heightR * width) + 2 * (heightR * depth) + 2 * (width * depth);

		int n = 0;
		while (yVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)aabbs.size() - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'y';
			splitPosition = split;
		}
	}

	//z axis
	min = fminf(zObjectMedian, zSpatialMedian);
	max = fmaxf(zObjectMedian, zSpatialMedian);
	step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float depthL = split - node->aabb.bounds[0].z;
		float depthR = node->aabb.bounds[1].z - split;

		float SALeft = 2 * (height * width) + 2 * (height * depthL) + 2 * (width * depthL);
		float SARight = 2 * (height * width) + 2 * (height * depthR) + 2 * (width * depthR);

		int n = 0;
		while (zVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)aabbs.size() - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'z';
			splitPosition = split;
		}
	}

	//if best cost is the same as the parent cost then don't split and just have a leaf node
	if (splitCost == parentCost) {
		return -1;
	}

	//return the first index for the right child as the point of partition
	split_position = splitPosition;

	switch (splitAxis) {
	case 'x':
		return 0;
		break;
	case 'y':
		return 1;
		break;
	case 'z':
		return 2;
		break;
	}
	return 0;
}

int count_nodes(BVHNode* root) {
	if (!root->is_leaf) {
		return 1 + count_nodes(root->left) + count_nodes(root->right);
	}

	return 1;
}

void BVH::PopulateCFBVH(unsigned int& cumulative_index, unsigned int& triangle_index, BVHNode* node) {
	int current_index = cumulative_index;
	this->cf_bvh[current_index].bounds[0] = node->aabb.bounds[0];
	this->cf_bvh[current_index].bounds[1] = node->aabb.bounds[1];

	if (!(node->is_leaf)) {
		//std::cout << "INNER " << cumulative_index << std::endl;
		cumulative_index++;
		unsigned int left_index = cumulative_index;
		PopulateCFBVH(cumulative_index, triangle_index, node->left);

		cumulative_index++;
		unsigned int right_index = cumulative_index;
		PopulateCFBVH(cumulative_index, triangle_index, node->right);

		cf_bvh[current_index].u.inner.index_left = left_index;
		cf_bvh[current_index].u.inner.index_right = right_index;
	}
	else {
		unsigned int count = (unsigned int)node->leaf_triangles.size();
		cf_bvh[current_index].u.leaf.count = 0x80000000 | count; //basically sets the first bit of count a
		cf_bvh[current_index].u.leaf.index_first_tri = triangle_index;
		//std::cout << "LEAF " << cumulative_index << std::endl;
		for (unsigned int i = 0; i < count; i++) {
			this->triangle_indices[triangle_index] = node->leaf_triangles[i];
			triangle_index++;
		}
	}
}

void BVH::ConstructCacheFriendly(int tri_count) {
	int node_count = count_nodes(this->root_node);

	this->cf_bvh = new BVHNode_CacheFriendly[node_count];
	this->triangle_indices = new int[tri_count];

	unsigned int cumulative_index = 0;
	unsigned int triangle_index = 0;
	PopulateCFBVH(cumulative_index, triangle_index, this->root_node);

	cudaAssert(Malloc(&(cf_bvh_gpu), node_count * sizeof(BVHNode_CacheFriendly)));
	cudaAssert(Malloc(&(triangle_indices_gpu), tri_count * sizeof(int)));
	cudaAssert(Memcpy(cf_bvh_gpu, cf_bvh, node_count * sizeof(BVHNode_CacheFriendly), hipMemcpyHostToDevice));
	cudaAssert(Memcpy(triangle_indices_gpu, triangle_indices, tri_count * sizeof(int), hipMemcpyHostToDevice));
}