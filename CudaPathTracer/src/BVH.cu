#include "hip/hip_runtime.h"
#pragma once
#include <precomp.h>

using namespace glm;
/////////////////////
/////BVH NODE //////
////////////////////

void BVHNode::CalculateBounds(std::vector<AABB> t_AABBs, int first, int count, int* indices, int tri_count) {
	if (tri_count <= 0) {
		return;
	}
	AABB aabb;
	aabb = t_AABBs[indices[first]];

	for (int i = first + 1; i < (first + count); i++) {
		if (aabb.bounds[1].x < t_AABBs[indices[i]].bounds[1].x) {
			aabb.bounds[1].x = t_AABBs[indices[i]].bounds[1].x;
		}

		if (aabb.bounds[0].x > t_AABBs[indices[i]].bounds[0].x) {
			aabb.bounds[0].x = t_AABBs[indices[i]].bounds[0].x;
		}

		if (aabb.bounds[1].y < t_AABBs[indices[i]].bounds[1].y) {
			aabb.bounds[1].y = t_AABBs[indices[i]].bounds[1].y;
		}

		if (aabb.bounds[0].y > t_AABBs[indices[i]].bounds[0].y) {
			aabb.bounds[0].y = t_AABBs[indices[i]].bounds[0].y;
		}

		if (aabb.bounds[1].z < t_AABBs[indices[i]].bounds[1].z) {
			aabb.bounds[1].z = t_AABBs[indices[i]].bounds[1].z;
		}

		if (aabb.bounds[0].z > t_AABBs[indices[i]].bounds[0].z) {
			aabb.bounds[0].z = t_AABBs[indices[i]].bounds[0].z;
		}
	}

	aabb.CalculateCentre();
	this->aabb = aabb;

	//printf("%f %f %f | %f %f %f \n", aabb.bounds[0].x, aabb.bounds[0].y, aabb.bounds[0].z, aabb.bounds[1].x, aabb.bounds[1].y, aabb.bounds[1].z);
}

/////////////
////AABB/////
/////////////
void AABB::CalculateCentre() {
	this->centre_point = vec3((bounds[0].x + bounds[1].x) / 2.0f, (bounds[0].y + bounds[1].y) / 2.0f, (bounds[0].z + bounds[1].z) / 2.0f);
}

///////////////
/////BVH //////
//////////////
BVH::BVH() {}

BVH::BVH(vec3* t_vertices_gpu, vec3* t_normals_gpu, int* t_indices_gpu) {
	this->t_vertices_gpu = t_vertices_gpu;
	this->t_normals_gpu = t_normals_gpu;
	this->t_indices_gpu = t_indices_gpu;
}

void BVH::ConstructBVH(std::vector<vec3> t_vertices, std::vector<int> t_indices, int tri_count) {
	this->t_AABBs = std::vector<AABB>();
	indices = new int[tri_count];


	for (int i = 0; i < tri_count; i++) {
		indices[i] = i;

		vec3 v1 = t_vertices[t_indices[i * 3 + 1]];
		vec3 v2 = t_vertices[t_indices[i * 3 + 2]];
		vec3 v3 = t_vertices[t_indices[i * 3]];

		AABB bounds = AABB();
		bounds.bounds[1].x = max(v1.x, max(v2.x, v3.x));
		bounds.bounds[0].x = min(v1.x, min(v2.x, v3.x));
		bounds.bounds[1].y = max(v1.y, max(v2.y, v3.y));
		bounds.bounds[0].y = min(v1.y, min(v2.y, v3.y));
		bounds.bounds[1].z = max(v1.z, max(v2.z, v3.z));
		bounds.bounds[0].z = min(v1.z, min(v2.z, v3.z));
		bounds.CalculateCentre();

		t_AABBs.push_back(bounds);
	}

	//create the root node of the tree
	this->root_node = new BVHNode();
	this->root_node->first = 0;
	this->root_node->count = tri_count;
	this->root_node->CalculateBounds(t_AABBs, this->root_node->first, this->root_node->count, indices, tri_count);

	//subdivide to create the child nodes recursively
	Subdivide(root_node, t_vertices, t_indices, tri_count);
}

void BVH::Subdivide(BVHNode* node, std::vector<vec3> t_vertices, std::vector<int> t_indices, int tri_count) {
	//base case, this is a leaf node if there are less than 3 objects inside
	if (node->count < 3) {
		node->is_leaf = true;
		for (int i = node->first; i < node->first + node->count; i++) {
			std::cout << i << " ";
		}
		std::cout << std::endl;
		return;
	}

	node->left = new BVHNode();
	node->right = new BVHNode();

	//calculate partition
	int split = Partition(node);
	//if no split was made because child nodes do not improve cost then make this a leaf
	if (split == -1) {
		node->is_leaf = true;
		for (int i = node->first; i < node->first + node->count; i++) {
			std::cout << i << " ";
		}
		std::cout << std::endl;
		return;
	}
	if (split == node->first) {
		return;
	}
	//else subdivide
	node->left->first = node->first;
	node->left->count = split - node->first;
	node->left->CalculateBounds(t_AABBs, node->left->first, node->left->count, indices, tri_count);

	node->right->first = split;
	node->right->count = node->count - node->left->count;
	node->right->CalculateBounds(t_AABBs, node->right->first, node->right->count, indices, tri_count);

	//recursive calls to partition child nodes
	if (node->left->count > 0) {
		Subdivide(node->left, t_vertices, t_indices, tri_count);
	}
	else {
		node->left->is_leaf = true;
	}
	if (node->right->count > 0) {
		Subdivide(node->right, t_vertices, t_indices, tri_count);
	}
	else {
		node->right->is_leaf = true;
	}

	node->is_leaf = false;

}

int BVH::Partition(BVHNode* node) {
	/*--FINDING SPLIT AXIS--*/
	//Using the Arbitrary Acyclic Algorithm from: http://graphicsinterface.org/wp-content/uploads/gi1989-22.pdf
	//spatial medians
	float xSpatialMedian = (node->aabb.bounds[0].x + node->aabb.bounds[1].x) / 2.0f;
	float ySpatialMedian = (node->aabb.bounds[0].y + node->aabb.bounds[1].y) / 2.0f;
	float zSpatialMedian = (node->aabb.bounds[0].z + node->aabb.bounds[1].z) / 2.0f;

	//object medians
	float xObjectMedian;
	float yObjectMedian;
	float zObjectMedian;

	float* xVals = new float[node->count];
	float* yVals = new float[node->count];
	float* zVals = new float[node->count];

	for (int i = node->first; i < (node->first + node->count); i++) {
		xVals[i - node->first] = t_AABBs[indices[i]].centre_point.x;
		yVals[i - node->first] = t_AABBs[indices[i]].centre_point.y;
		zVals[i - node->first] = t_AABBs[indices[i]].centre_point.z;
	}

	std::sort(xVals, xVals + node->count);
	std::sort(yVals, yVals + node->count);
	std::sort(zVals, zVals + node->count);

	xObjectMedian = xVals[node->count / 2];
	yObjectMedian = yVals[node->count / 2];
	zObjectMedian = zVals[node->count / 2];

	//choose 9 equally spaced possibilities on each axis and check cost.
	float height = node->aabb.bounds[1].y - node->aabb.bounds[0].y;
	float width = node->aabb.bounds[1].x - node->aabb.bounds[0].x;
	float depth = node->aabb.bounds[1].z - node->aabb.bounds[0].z;

	float SA = 2 * (height * width) + 2 * (height * depth) + 2 * (width * depth);
	float parentCost = SA * node->count;
	float splitCost = parentCost;
	char splitAxis = 'x';
	float splitPosition = xSpatialMedian;
	//x axis
	float min = fminf(xObjectMedian, xSpatialMedian);
	float max = fmaxf(xObjectMedian, xSpatialMedian);
	float step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float widthL = split - node->aabb.bounds[0].x;
		float widthR = node->aabb.bounds[1].x - split;

		float SALeft = 2 * (height * widthL) + 2 * (height * depth) + 2 * (widthL * depth);
		float SARight = 2 * (height * widthR) + 2 * (height * depth) + 2 * (widthR * depth);

		int n = 0;
		while (xVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)node->count - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'x';
			splitPosition = split;
		}
	}

	//y axis
	min = fminf(yObjectMedian, ySpatialMedian);
	max = fmaxf(yObjectMedian, ySpatialMedian);
	step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float heightL = split - node->aabb.bounds[0].y;
		float heightR = node->aabb.bounds[1].y - split;

		float SALeft = 2 * (heightL * width) + 2 * (heightL * depth) + 2 * (width * depth);
		float SARight = 2 * (heightR * width) + 2 * (heightR * depth) + 2 * (width * depth);

		int n = 0;
		while (yVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)node->count - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'y';
			splitPosition = split;
		}
	}

	//z axis
	min = fminf(zObjectMedian, zSpatialMedian);
	max = fmaxf(zObjectMedian, zSpatialMedian);
	step = (max - min) / 9.0f;
	for (int i = 0; i < 9; i++) {
		float split = (min + (step * i));
		float tempCost;

		float depthL = split - node->aabb.bounds[0].z;
		float depthR = node->aabb.bounds[1].z - split;

		float SALeft = 2 * (height * width) + 2 * (height * depthL) + 2 * (width * depthL);
		float SARight = 2 * (height * width) + 2 * (height * depthR) + 2 * (width * depthR);

		int n = 0;
		while (zVals[n] < split) {
			n++;
		}

		tempCost = (SALeft * (float)n) + (SARight * ((float)node->count - (float)n));
		if (tempCost < splitCost) {
			splitCost = tempCost;
			splitAxis = 'z';
			splitPosition = split;
		}
	}

	//if best cost is the same as the parent cost then don't split and just have a leaf node
	if (splitCost == parentCost){
		return -1;
	}
	std::vector<int> lessEqual;
	std::vector<int> more;
	//sort into 2 arrays of object indices on the left and right of partition
	for (int i = node->first; i < (node->first + node->count); i++) {
		switch (splitAxis) {
		case 'x':
			if (t_AABBs[indices[i]].centre_point.x <= splitPosition) {
				lessEqual.push_back(indices[i]);
			}
			else {
				more.push_back(indices[i]);
			}
			break;
		case 'y':
			if (t_AABBs[indices[i]].centre_point.y <= splitPosition) {
				lessEqual.push_back(indices[i]);
			}
			else {
				more.push_back(indices[i]);
			}
			break;
		case 'z':
			if (t_AABBs[indices[i]].centre_point.z <= splitPosition) {
				lessEqual.push_back(indices[i]);
			}
			else {
				more.push_back(indices[i]);
			}
			break;
		}
	}

	//if next partition is same as current partition
	if (lessEqual.size() == 0 || more.size() == 0) {
		return -1;
	}

	//rearrange indices according to partition
	int partitionPoint = node->first + node->count - 1;
	for (int i = 0; i < lessEqual.size(); i++) {
		indices[node->first + i] = lessEqual[i];
	}
	for (int i = 0; i < more.size(); i++) {
		indices[node->first + lessEqual.size() + i] = more[i];
	}
	//return the first index for the right child as the point of partition
	return node->first + lessEqual.size();
}