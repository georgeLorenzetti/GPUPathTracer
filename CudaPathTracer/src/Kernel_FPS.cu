#include "hip/hip_runtime.h"
#pragma once
#include <precomp.h>
#include <surface_functions.h>
using namespace glm;

//kernel variables
__device__ int counter_primary = 0;
__device__ int counter_extend = 0;
__device__ int counter_shade = 0;
__device__ int counter_connect = 0;
__device__ int start_position = 0;
__device__ int count_shadow_ray = 0;
__device__ int connect_ray_index = 0;

//debug variables
__device__ int debug_count = 0;
__device__ int y_count = 0;
__device__ bool set = false;

//draw variables
surface<void, cudaSurfaceType2D> screen;


/**DEVICE FUNCTIONS**/
//get random int
__device__ unsigned int random_int(unsigned int& seed) {
	seed ^= seed << 13;
	seed ^= seed << 17;
	seed ^= seed << 5;
	return seed;
}

//get random float
__device__ float random_float(unsigned int& seed) {
	return float(random_int(seed)) * 2.3283064365387e-10f;
}

__device__ vec3 cosineWeightedSample(const vec3& normal, unsigned int& seed) {
	float rand1 = random_float(seed);
	float rand2 = random_float(seed);

	const float r = sqrtf(rand1);
	const float theta = DOUBLEPI * rand2;

	const float x = r * cosf(theta);
	const float z = r * sinf(theta);

	vec3 sampleDirection = vec3(x, sqrtf(max(0.0f, 1 - rand1)), z);
	sampleDirection = normalize(sampleDirection);
	vec3 Nb;
	vec3 Nt;

	//createing coordinate systen around the normal
	if (fabs(normal.x) > fabs(normal.y)) {
		float denom = 1 / sqrtf(normal.x * normal.x + normal.z * normal.z);
		Nt = vec3(normal.z, 0, -1.0f * normal.x) * denom;
	}
	else {
		float denom = 1 / sqrtf(normal.y * normal.y + normal.z * normal.z);
		Nt = vec3(0, -1.0f * normal.z, normal.y) * denom;
	}
	Nb = cross(normal, Nt);

	Nt = vec3(normal.y * Nb.z - normal.z * Nb.y, normal.z * Nb.x - normal.x * Nb.z, normal.x * Nb.y - normal.y * Nb.x);

	vec3 result = vec3(sampleDirection.x * Nb.x + sampleDirection.y * normal.x + sampleDirection.z * Nt.x, sampleDirection.x * Nb.y + sampleDirection.y * normal.y + sampleDirection.z * Nt.y, sampleDirection.x * Nb.z + sampleDirection.y * normal.z + sampleDirection.z * Nt.z);
	result = normalize(result);
	return result;
}

//take uniform random sample from hemisphere
__device__ vec3 sampleHemisphere(const vec3& normal, unsigned int& seed) {
	float rand1 = random_float(seed);
	float rand2 = random_float(seed);

	float sinTheta = sqrtf(1 - rand1 * rand1);
	float phi = DOUBLEPI * rand2;
	float x = sinTheta * cosf(phi);
	float z = sinTheta * sinf(phi);
	vec3 sampleDirection = vec3(x, rand1, z);
	vec3 Nb;
	vec3 Nt;

	//creating coordinate systen around the normal
	if (fabs(normal.x) > fabs(normal.y)) {
		float denom = 1 / sqrtf(normal.x * normal.x + normal.z * normal.z);
		Nt = vec3(normal.z, 0, -1.0f * normal.x) * denom;
	}
	else {
		float denom = 1 / sqrtf(normal.y * normal.y + normal.z * normal.z);
		Nt = vec3(0, -1.0f * normal.z, normal.y) * denom;
	}
	Nb = cross(normal, Nt);

	Nt = vec3(normal.y * Nb.z - normal.z * Nb.y, normal.z * Nb.x - normal.x * Nb.z, normal.x * Nb.y - normal.y * Nb.x);

	vec3 result = vec3(sampleDirection.x * Nb.x + sampleDirection.y * normal.x + sampleDirection.z * Nt.x, sampleDirection.x * Nb.y + sampleDirection.y * normal.y + sampleDirection.z * Nt.y, sampleDirection.x * Nb.z + sampleDirection.y * normal.z + sampleDirection.z * Nt.z);
	result = normalize(result);
	return result;
}

//triangle intersection
__device__ bool intersect_triangle(const Ray& ray, const vec3& v0, const vec3& v1, const vec3& v2, vec3& intersection_point, float& t) {
	const float epsilon = 0.0000001;
	vec3 edge1, edge2, h, s, q;
	float a, f, u, v;
	edge1 = v1 - v0;
	edge2 = v2 - v0;
	h = cross(ray.direction, edge2);
	a = dot(edge1, h);

	if (a > -epsilon && a < epsilon)
		return false;

	f = 1.0 / a;
	s = ray.origin - v0;
	u = f * (dot(s, h));
	if (u < 0.0 || u > 1.0)
		return false;

	q = cross(s, edge1);
	v = f * dot(ray.direction, q);
	if (v < 0.0 || u + v > 1.0)
		return false;

	t = f * dot(edge2, q);
	intersection_point = ray.origin + ray.direction * t;
	if (t > epsilon) {
		return true;
	}
	else
		return false;
}

__device__ bool intersect_AABB(const vec3& origin, const vec3& direction_inverse, const vec3& aabb_min, const vec3& aabb_max) {
	vec3 t1 = (aabb_min - origin) * direction_inverse;
	vec3 t2 = (aabb_max - origin) * direction_inverse;

	vec3 min = glm::min(t1, t2);
	vec3 max = glm::max(t1, t2);

	float tmin = glm::max(min.x, glm::max(min.y, min.z));
	float tmax = glm::min(max.x, glm::min(max.y, max.z));

	return (tmax >= 0.0f && tmin < tmax);
}

__device__ int* intersect_MBVH_node(const vec3& origin, const vec3& direction, const vec3& direction_inverse, const float& t,
	const vec4& bounds_min_x, const vec4& bounds_max_x,
	const vec4& bounds_min_y, const vec4& bounds_max_y,
	const vec4& bounds_min_z, const vec4& bounds_max_z,
	bvec4& result) {

	union { vec4 t_min; int t_min_i[4]; };
	vec4 t1 = (bounds_min_x - origin.x) * direction_inverse.x;
	vec4 t2 = (bounds_max_x - origin.x) * direction_inverse.x;

	t_min = glm::min(t1, t2);
	vec4 t_max = glm::max(t1, t2);

	t1 = (bounds_min_y - origin.y) * direction_inverse.y;
	t2 = (bounds_max_y - origin.y) * direction_inverse.y;

	t_min = glm::max(t_min, glm::min(t1, t2));
	t_max = glm::min(t_max, glm::max(t1, t2));

	t1 = (bounds_min_z - origin.z) * direction_inverse.z;
	t2 = (bounds_max_z - origin.z) * direction_inverse.z;

	t_min = glm::max(t_min, glm::min(t1, t2));
	t_max = glm::min(t_max, glm::max(t1, t2));

	t_min_i[0] = ((t_min_i[0] & 0xFFFFFFFC) | 0b00);
	t_min_i[1] = ((t_min_i[1] & 0xFFFFFFFC) | 0b01);
	t_min_i[2] = ((t_min_i[2] & 0xFFFFFFFC) | 0b10);
	t_min_i[3] = ((t_min_i[3] & 0xFFFFFFFC) | 0b11);

	result[0] = (t_max[0] > 0.0f) && (t_min[0] <= t_max[0]) && (t_min[0] < t);
	result[1] = (t_max[1] > 0.0f) && (t_min[1] <= t_max[1]) && (t_min[1] < t);
	result[2] = (t_max[2] > 0.0f) && (t_min[2] <= t_max[2]) && (t_min[2] < t);
	result[3] = (t_max[3] > 0.0f) && (t_min[3] <= t_max[3]) && (t_min[3] < t);

	if (t_min[0] > t_min[1])
		swap(t_min[0], t_min[1]);
	if (t_min[2] > t_min[3])
		swap(t_min[2], t_min[3]);
	if (t_min[0] > t_min[2])
		swap(t_min[0], t_min[2]);
	if (t_min[1] > t_min[3])
		swap(t_min[1], t_min[3]);
	if (t_min[2] > t_min[3])
		swap(t_min[2], t_min[3]);

	return t_min_i;
}

__device__ bool shadow_traverse_BVH(const vec3* t_vertices_gpu, const vec3* t_normals_gpu, const Material* t_mats_gpu, int* t_indices_gpu, Ray* ray, BVHNode_CacheFriendly* bvh, int* bvh_tri_list) {

	int best_index = -1;
	float shortest_distance;

	// start from infinity
	shortest_distance = FLT_MAX;

	// create a stack for each ray
	// the stack is just a fixed size array of indices to BVH nodes
	int access_stack[64];

	int stack_index = 0;
	access_stack[0] = 0;
	stack_index++;

	vec3 intersection_point;

	// while the stack is not empty
	while (stack_index) {

		//pop from stack
		BVHNode_CacheFriendly current_node = bvh[access_stack[stack_index - 1]];
		stack_index--;

		if (!(current_node.u.leaf.count & 0x80000000)) { // if inner node
			if (intersect_AABB(ray->origin, (1.0f / ray->direction), current_node.bounds[0], current_node.bounds[1])) {
				access_stack[stack_index] = current_node.u.inner.index_right;
				stack_index++;
				access_stack[stack_index] = current_node.u.inner.index_left;
				stack_index++;
				if (stack_index > 64) {
					return false;
				}
			}
		}
		else { //else if leaf node
			for (int i = current_node.u.leaf.index_first_tri; i < current_node.u.leaf.index_first_tri + (current_node.u.leaf.count & 0x7fffffff); i++) {
				float current_t;
				vec3 intersection_point;

				bool intersected_something = intersect_triangle(*ray,
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3]],
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3 + 1]],
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3 + 2]],
					intersection_point,
					current_t);

				if (!intersected_something || current_t < 0 || current_t >= ray->t) {
					continue;
				}

				ray->t = current_t;
				ray->intersection_point = intersection_point;

				vec3 normal = t_normals_gpu[bvh_tri_list[i]];
				float dot_product = dot(ray->direction, normal);
				if (dot_product > 0.0f) {
					normal *= -1.0f;
				}

				ray->t = current_t;
				ray->intersected_material = t_mats_gpu[bvh_tri_list[i]];
			}
		}

	}

	return (best_index != -1);
}

__device__ bool shadow_traverse_MBVH(const vec3* t_vertices_gpu, const vec3* t_normals_gpu, const Material* t_mats_gpu, int* t_indices_gpu, Ray* ray, MBVHNode_CacheFriendly* mbvh, int* mbvh_tri_list) {

	int best_index = -1;
	float shortest_distance;

	// start from infinity
	shortest_distance = FLT_MAX;

	// create a stack for each ray
	// the stack is just a fixed size array of indices to BVH nodes
	int access_stack[64];

	int stack_index = 0;
	access_stack[0] = 0;
	stack_index++;

	vec3 intersection_point;

	// while the stack is not empty
	while (stack_index) {

		//pop from stack
		MBVHNode_CacheFriendly current_node = mbvh[access_stack[stack_index - 1]];
		stack_index--;

		if (!(current_node.u.leaf.count & 0x80000000)) { // if inner node

			//check all children for AABB collisions
			vec4 bounds_min_x = vec4(0.0f);
			vec4 bounds_max_x = vec4(0.0f);
			vec4 bounds_min_y = vec4(0.0f);
			vec4 bounds_max_y = vec4(0.0f);
			vec4 bounds_min_z = vec4(0.0f);
			vec4 bounds_max_z = vec4(0.0f);

			for (int i = 0; i < current_node.u.inner.child_count; i++) {
				bounds_min_x[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].x;
				bounds_max_x[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].x;

				bounds_min_y[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].y;
				bounds_max_y[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].y;

				bounds_min_z[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].z;
				bounds_max_z[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].z;
			}
			bvec4 result = bvec4();
			int* t_min_i = intersect_MBVH_node(ray->origin, ray->direction, 1.0f / ray->direction, ray->t, bounds_min_x, bounds_max_x, bounds_min_y, bounds_max_y, bounds_min_z, bounds_max_z, result);

			//add the hits to the stack
			if (any(result)) {
				for (int i = current_node.u.inner.child_count - 1; i >= 0; i--) {
					const int idx = (t_min_i[i] & 0b11);
					if (result[i] == 1) {
						access_stack[stack_index] = current_node.u.inner.child_index + i;

						stack_index++;
						if (stack_index > 64) {
							return false;
						}
					}
				}
			}
		}
		else { //else if leaf node
			for (int i = current_node.u.leaf.index_first_tri; i < current_node.u.leaf.index_first_tri + (current_node.u.leaf.count & 0x7fffffff); i++) {
				float current_t;
				vec3 intersection_point;

				bool intersected_something = intersect_triangle(*ray,
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3]],
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3 + 1]],
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3 + 2]],
					intersection_point,
					current_t);

				if (!intersected_something || current_t < 0 || current_t >= ray->t) {
					continue;
				}

				ray->t = current_t;
				ray->intersection_point = intersection_point;

				vec3 normal = t_normals_gpu[mbvh_tri_list[i]];
				float dot_product = dot(ray->direction, normal);
				if (dot_product > 0.0f) {
					normal *= -1.0f;
				}

				ray->t = current_t;
				ray->intersected_material = t_mats_gpu[mbvh_tri_list[i]];
			}
		}

	}
	return (best_index != -1);
}

__device__ bool traverse_MBVH(vec3* t_vertices_gpu, vec3* t_normals_gpu, int* t_indices_gpu, Ray* ray, MBVHNode_CacheFriendly* mbvh, int* mbvh_tri_list, unsigned int seed) {

	int best_index = -1;
	float shortest_distance;

	//start from infinity
	shortest_distance = FLT_MAX;

	//create a stack of each ray
	//the stack is just a fixed size array of indices to BVH nodes
	int access_stack[64];

	int stack_index = 0;
	access_stack[0] = 0;
	stack_index++;

	vec3 intersection_point;

	// while the stack is not empty
	while (stack_index) {
		//pop from stack
		MBVHNode_CacheFriendly current_node = mbvh[access_stack[stack_index - 1]];
		stack_index--;

		if (!(current_node.u.leaf.count & 0x80000000)) { // if inner node
			//check all children for AABB collisions
			vec4 bounds_min_x = vec4(0.0f);
			vec4 bounds_max_x = vec4(0.0f);
			vec4 bounds_min_y = vec4(0.0f);
			vec4 bounds_max_y = vec4(0.0f);
			vec4 bounds_min_z = vec4(0.0f);
			vec4 bounds_max_z = vec4(0.0f);

			for (int i = 0; i < current_node.u.inner.child_count; i++) {
				bounds_min_x[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].x;
				bounds_max_x[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].x;

				bounds_min_y[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].y;
				bounds_max_y[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].y;

				bounds_min_z[i] = mbvh[current_node.u.inner.child_index + i].bounds[0].z;
				bounds_max_z[i] = mbvh[current_node.u.inner.child_index + i].bounds[1].z;
			}
			bvec4 result = bvec4();
			int* t_min_i = intersect_MBVH_node(ray->origin, ray->direction, 1.0f / ray->direction, ray->t, bounds_min_x, bounds_max_x, bounds_min_y, bounds_max_y, bounds_min_z, bounds_max_z, result);

			//add the hits to the stack
			if (any(result)) {
				for (int i = current_node.u.inner.child_count - 1; i >= 0; i--) {
					const int idx = (t_min_i[i] & 0b11);
					if (result[i] == 1) {
						access_stack[stack_index] = current_node.u.inner.child_index + i;
						stack_index++;
						if (stack_index > 64) {
							return false;
						}
					}
				}
			}
		}
		else { //else if leaf node
			for (int i = current_node.u.leaf.index_first_tri; i < current_node.u.leaf.index_first_tri + (current_node.u.leaf.count & 0x7fffffff); i++) {
				float current_t;
				vec3 intersection_point;

				bool intersected_something = intersect_triangle(*ray,
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3]],
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3 + 1]],
					t_vertices_gpu[t_indices_gpu[mbvh_tri_list[i] * 3 + 2]],
					intersection_point,
					current_t);

				if (!intersected_something || current_t < 0 || current_t >= ray->t) {
					continue;
				}

				ray->t = current_t;
				ray->intersection_point = intersection_point;

				vec3 normal = t_normals_gpu[mbvh_tri_list[i]];
				float dot_product = dot(ray->direction, normal);
				if (dot_product > 0.0f) {
					normal *= -1.0f;
				}

				ray->reflected_direction = normalize(cosineWeightedSample(normal, seed));
				ray->intersection_index = mbvh_tri_list[i];
				best_index = mbvh_tri_list[i];
			}
		}
	}
	return (best_index != -1);
}

__device__ bool traverse_BVH(vec3* t_vertices_gpu, vec3* t_normals_gpu, int* t_indices_gpu, Ray* ray, BVHNode_CacheFriendly* bvh, int* bvh_tri_list, unsigned int seed, int rayN, int debug = false) {
	int best_index = -1;
	float shortest_distance;

	// start from infinity
	shortest_distance = FLT_MAX;

	// create a stack for each ray
	// the stack is just a fixed size array of indices to BVH nodes
	int access_stack[64];

	int stack_index = 0;
	access_stack[0] = 0;
	stack_index++;

	vec3 intersection_point;

	// while the stack is not empty
	while (stack_index) {
		//pop from stack
		BVHNode_CacheFriendly current_node = bvh[access_stack[stack_index - 1]];
		stack_index--;

		if (!(current_node.u.leaf.count & 0x80000000)) { // if inner node
			if (intersect_AABB(ray->origin, (1.0f / ray->direction), current_node.bounds[0], current_node.bounds[1])) {
				access_stack[stack_index] = current_node.u.inner.index_right;
				stack_index++;
				access_stack[stack_index] = current_node.u.inner.index_left;
				stack_index++;
				if (stack_index > 64) {
					return false;
				}
			}
		}
		else { //else if leaf node
			for (int i = current_node.u.leaf.index_first_tri; i < current_node.u.leaf.index_first_tri + (current_node.u.leaf.count & 0x7fffffff); i++) {
				float current_t;
				vec3 intersection_point;

				bool intersected_something = intersect_triangle(*ray,
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3]],
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3 + 1]],
					t_vertices_gpu[t_indices_gpu[bvh_tri_list[i] * 3 + 2]],
					intersection_point,
					current_t);

				if (!intersected_something || current_t < 0 || current_t >= ray->t) {
					continue;
				}

				ray->t = current_t;
				ray->intersection_point = intersection_point;

				vec3 normal = t_normals_gpu[bvh_tri_list[i]];
				float dot_product = dot(ray->direction, normal);
				if (dot_product > 0.0f) {
					normal *= -1.0f;
				}
				ray->reflected_direction = normalize(cosineWeightedSample(normal, seed));
				ray->intersection_index = bvh_tri_list[i];
				best_index = bvh_tri_list[i];
			}
		}

	}

	return (best_index != -1);
}

__device__ vec3 GetTextureColour(int index, float u, float v, vec3* texture_buffer, vec3* texture_descriptors) {
	int width = texture_descriptors[index].g;
	int height = texture_descriptors[index].b;

	float x = fmod(u, 1.0f);
	float y = fmod(v, 1.0f);

	if (x < 0) x += 1.0f;
	if (y < 0) y += 1.0f;

	int ix = int(x * (width - 1));
	int iy = int(y * (height - 1));

	int tex_buffer_index = texture_descriptors[index].r + (ix + iy * width);
	return texture_buffer[tex_buffer_index];
}
//draw to cuda surface
__device__ void Draw(vec4& colour, int x, int y) {
	surf2Dwrite(colour, screen, x * sizeof(vec4), y);
}

/**DEBUG KERNELS/FUNCTIONS**/
//generic setup for printing things
__global__ void print_helper(const Scene scene, MBVHNode_CacheFriendly* mbvh, int* mbvh_tri_list, int frame) {
	printf("frame\n");
}

/**MAIN KERNELS**/
//reset kernel variables
__global__ void ResetAllVariables() {
	counter_primary = 0;
	counter_extend = 0;
	counter_shade = 0;
	counter_connect = 0;
	start_position = 0;
	count_shadow_ray = 0;
	connect_ray_index = 0;
	debug_count = 0;
}

__global__ void SetGlobalVariables(int ray_buffer_size) {

	const unsigned int last_frame_stop = ray_buffer_size - connect_ray_index;
	start_position += last_frame_stop;
	start_position = start_position % (SCRWIDTH * SCRHEIGHT);

	counter_primary = 0;
	counter_extend = 0;
	counter_shade = 0;
	counter_connect = 0;
	connect_ray_index = 0;
	debug_count = 0;
	y_count = 0;
	count_shadow_ray = 0;
}

//process and draw each pixel colour
__global__ void draw_frame(vec4* frame_buffer, const Scene scene) {

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= SCRWIDTH || y >= SCRHEIGHT) {
		return;
	}

	const int index = x + (y * SCRWIDTH);
	vec3 temp_colour = vec3();

	//sample counter is stored in .a 
	temp_colour.r = frame_buffer[index].r / frame_buffer[index].a;
	temp_colour.g = frame_buffer[index].g / frame_buffer[index].a;
	temp_colour.b = frame_buffer[index].b / frame_buffer[index].a;

	vec3 exponent = vec3(1.0f / 2.2f);
	vec4 colour = vec4(pow(temp_colour, exponent), 1.0f);
	Draw(colour, x, y);
}

//genereate kernel
__global__ void GeneratePrimaryRays(const Scene& scene, const vec3 topLeft, const vec3 stepH, const vec3 stepV, const  vec3 c_position, Ray* ray_buffer, int ray_buffer_size, const int frame, vec4* frame_buffer) {

	while (true) {
		int index = atomicAdd(&counter_primary, 1);
		int buffer_index = index + connect_ray_index;

		if (buffer_index >= ray_buffer_size) {
			return;
		}


		const int x = (start_position + index) % SCRWIDTH;
		const int y = ((start_position + index) / SCRWIDTH) % SCRHEIGHT;

		vec3 pixelPoint = vec3(topLeft.x, topLeft.y, topLeft.z) + (stepV * (y + 0.5f)) + (stepH * (x + 0.5f));
		vec3 rayDirection = vec3(pixelPoint.x - c_position.x, pixelPoint.y - c_position.y, pixelPoint.z - c_position.z);
		vec3 rayOrigin = vec3(c_position.x, c_position.y, c_position.z);
		rayDirection = normalize(rayDirection);

		
		ray_buffer[buffer_index] = Ray(rayOrigin, rayDirection, x  + (y * SCRWIDTH));

		atomicAdd(&(frame_buffer[ray_buffer[buffer_index].pixel_index].a), 1.0f);
	}
}

//extend kernel
__global__ void Extend(const Scene scene, MBVHNode_CacheFriendly* mbvh, int* mbvh_tri_list, Ray* ray_buffer, int ray_buffer_size, int triangle_count, int frame) {
	while (true) {
		int index = atomicAdd(&counter_extend, 1);
		unsigned int seed = (index + frame * 147565741) * 720898027 * index;
		if (index >= ray_buffer_size) {
			return;
		}

		bool hit = traverse_MBVH(scene.t_vertices_gpu, scene.t_normals_gpu, scene.t_indices_gpu, &ray_buffer[index], mbvh, mbvh_tri_list, seed);
	}
}

//shade kernel
__global__ void Shade(const Scene scene, Ray* shadow_ray_buffer, Ray* ray_buffer, Ray* ray_buffer_next, int ray_buffer_size, int triangle_count, int frame, vec4* frame_buffer) {
	while (true) {
		int index = atomicAdd(&counter_shade, 1);
		unsigned int seed = (index + frame * 147565741) * 720898027 * index;

		if (index >= ray_buffer_size) {
			return;
		}

		Ray* current_ray = &ray_buffer[index];
		float max_distance = MAXDISTANCE;
		if (ray_buffer[index].t < max_distance && ray_buffer[index].bounce <= MAXBOUNCE) {
			current_ray->intersected_material = scene.t_mats_gpu[current_ray->intersection_index];
		}

		//check material, if specular then decide if this ray is a diffuse or reflected
		if (current_ray->intersected_material.type == 3) {
			float r = random_float(seed);
			if (r > current_ray->intersected_material.specularity) {
				current_ray->intersected_material.type = 2;
			}
		}

		vec3 normal;
		switch (current_ray->intersected_material.type) {
			//Background
		case 0:
			current_ray->cumulative_colour = vec3(0);
			if (current_ray->bounce == 0 || current_ray->last_specular) {
				vec2 uv = { 1.0f + atan2f(current_ray->direction.x, -current_ray->direction.z) * glm::one_over_pi<float>() * 0.5f, 1.0f - acosf(current_ray->direction.y) * glm::one_over_pi<float>() };
				int index = uv.x + (uv.y * SCRWIDTH);
				vec3 skybox_colour = GetTextureColour(0, uv.x, uv.y, scene.texture_buffer_gpu, scene.texture_descriptors_gpu);
				current_ray->cumulative_colour = skybox_colour;
			}

			current_ray->terminate_flag = true;
			break;
			//Light
		case 1:
			current_ray->cumulative_colour = vec3(0);
			if (current_ray->bounce == 0 || current_ray->last_specular) {
				current_ray->cumulative_colour = vec3(1.0f, 1.0f, 1.0f);
			}
			current_ray->terminate_flag = true;
			break;
			//Labertian
		case 2:
			float rand = random_float(seed);
			float split = 0;

			int counter = 1;
			vec3 random_point = vec3(0.0f);
			while (counter <= scene.light_tri_count) {
				split += scene.light_areas_gpu[counter - 1];
				float proportion = split / scene.total_light_area;

				if (proportion > rand) {

					//get random point on the light
					vec3 va = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3)]];
					vec3 vb = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3) + 1]];
					vec3 vc = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3) + 2]];
					vec3 ab = vb - va;
					vec3 ac = vc - va;

					float w1 = random_float(seed);
					float w2 = random_float(seed);

					random_point = va + (w1 * ab) + (w2 * ac);
					break;
				}
				counter++;
			}

			normal = scene.t_normals_gpu[current_ray->intersection_index];
			if (dot(current_ray->reflected_direction, normal) < 0.0f) {
				normal *= -1.0f;
			}

			vec3 BRDF = vec3(current_ray->intersected_material.colour.r * INVPI, current_ray->intersected_material.colour.g * INVPI, current_ray->intersected_material.colour.b * INVPI);
			float inv_pdf_hemisphere_sample = PI / dot(current_ray->reflected_direction, normal);

			vec3 shadow_ray_direction = random_point - current_ray->intersection_point;
			float distance_sqared = dot(shadow_ray_direction, shadow_ray_direction);
			shadow_ray_direction = normalize(shadow_ray_direction);
			vec3 shadow_ray_origin = current_ray->intersection_point + (0.0001f * shadow_ray_direction);

			vec3 light_normal = scene.t_normals_gpu[scene.tri_count - counter];
			if (dot(-1.0f * shadow_ray_direction, light_normal) < 0.0f) {
				light_normal *= -1.0f;
			}

			float n_dot_l = dot(normal, shadow_ray_direction);
			float ln_dot_l = dot(light_normal, -1.0f * shadow_ray_direction);
			if (ln_dot_l > 0 && n_dot_l > 0) {

				float area = scene.light_areas_gpu[counter - 1];
				float inverse_area_pdf = scene.total_light_area / area;
				float solid_angle = (area * (ln_dot_l)) / distance_sqared;

				float pdf1 = 1 / solid_angle;
				float pdf2 = 1 / inv_pdf_hemisphere_sample;

				float combined_pdf = ((pdf1 / (pdf1 + pdf2)) * pdf1) + ((pdf2 / (pdf1 + pdf2)) * pdf2);


				vec3 shadow_colour = BRDF * scene.emission * inverse_area_pdf * solid_angle * n_dot_l * current_ray->cumulative_colour;
				int shadow_index = atomicAdd(&count_shadow_ray, 1);
				shadow_ray_buffer[shadow_index] = Ray(shadow_ray_origin, shadow_ray_direction, current_ray->pixel_index, shadow_colour);
			}
			vec3 addition = inv_pdf_hemisphere_sample * BRDF * (dot(current_ray->reflected_direction, normal));
			current_ray->cumulative_colour *= addition;
			current_ray->last_specular = false;
			break;
		case 3:
			normal = scene.t_normals_gpu[current_ray->intersection_index];
			if (dot(current_ray->reflected_direction, normal) < 0.0f) {
				normal *= -1.0f;
			}
			current_ray->reflected_direction = reflect(current_ray->direction, normal);
			current_ray->last_specular = true;
			break;
		case 4:
			normal = scene.t_normals_gpu[current_ray->intersection_index];
			if (dot(current_ray->reflected_direction, normal) < 0.0f) {
				normal *= -1.0f;
			}
			current_ray->reflected_direction = refract(current_ray->direction, normal, current_ray->intersected_material.specularity);
			current_ray->last_specular = true;
		default:
			break;
		}

		switch (current_ray->terminate_flag) {
		case true:
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);
			break;
		case false:
			vec3 ray_origin = current_ray->intersection_point + (current_ray->reflected_direction * 0.00001f);
			int e_index = atomicAdd(&connect_ray_index, 1);
			ray_buffer_next[e_index] = Ray(ray_origin, current_ray->reflected_direction, current_ray->pixel_index, current_ray->cumulative_colour);
			ray_buffer_next[e_index].bounce = current_ray->bounce + 1;
			ray_buffer_next[e_index].last_specular = current_ray->last_specular;
			break;
		default:
			break;
		}
	}
}

__global__ void ShadeReference(const Scene& scene, Ray* ray_buffer, Ray* ray_buffer_next, int ray_buffer_size, vec4* frame_buffer) {
	while (true) {
		int index = atomicAdd(&counter_shade, 1);

		if (index >= ray_buffer_size) {
			return;
		}
		Ray* current_ray = &ray_buffer[index];
		float max_distance = MAXDISTANCE;

		if (ray_buffer[index].t < max_distance && ray_buffer[index].bounce <= MAXBOUNCE) {
			current_ray->intersected_material = scene.t_mats_gpu[current_ray->intersection_index];
		}

		switch (current_ray->intersected_material.type) {
			//Background
		case 0:
			if (current_ray->bounce == 0) {
				current_ray->cumulative_colour = scene.bg_colour;
			}
			else {
				current_ray->cumulative_colour = vec3(0.0f, 0.0f, 0.0f);
			}

			current_ray->terminate_flag = true;
			break;
			//Light
		case 1:
			current_ray->cumulative_colour = current_ray->cumulative_colour * scene.emission;
			current_ray->terminate_flag = true;
			break;
			//Labertian
		case 2:
			vec3 normal = scene.t_normals_gpu[current_ray->intersection_index];
			float dot_product = dot(current_ray->reflected_direction, normal);
			if (dot_product < 0.0f) {
				normal *= -1.0f;
				dot_product = dot(current_ray->reflected_direction, normal);
			}
			vec3 BRDF = vec3(current_ray->intersected_material.colour.r * INVPI, current_ray->intersected_material.colour.g * INVPI, current_ray->intersected_material.colour.b * INVPI);
			float inv_PDF = PI / dot(current_ray->reflected_direction, normal);

			vec3 addition = inv_PDF * BRDF * dot_product;
			current_ray->cumulative_colour *= addition;
			break;
		default:
			break;
		}

		switch (current_ray->terminate_flag) {
		case true:
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);

			atomicAdd(&debug_count, 1);

			break;
		case false:
			vec3 ray_origin = current_ray->intersection_point + (current_ray->reflected_direction * 0.00001f);
			int e_index = atomicAdd(&connect_ray_index, 1);
			ray_buffer_next[e_index] = Ray(ray_origin, current_ray->reflected_direction, current_ray->pixel_index, current_ray->cumulative_colour);
			ray_buffer_next[e_index].bounce = current_ray->bounce + 1;
			break;
		default:
			break;
		}
	}
}

//connect kernel
__global__ void Connect(const Scene scene, MBVHNode_CacheFriendly* bvh, int* bvh_tri_list, Ray* shadow_ray_buffer, int triangle_count, vec4* frame_buffer) {
	while (true) {
		int index = atomicAdd(&counter_connect, 1);

		if (index >= count_shadow_ray) {
			return;
		}

		Ray* current_ray = &shadow_ray_buffer[index];

#if 1
		shadow_traverse_MBVH(scene.t_vertices_gpu, scene.t_normals_gpu, scene.t_mats_gpu, scene.t_indices_gpu, &shadow_ray_buffer[index], bvh, bvh_tri_list);
#else
		for (int i = 0; i < triangle_count; i++) {
			float current_t;
			vec3 intersection_point;

			bool intersected_something = intersect_triangle(shadow_ray_buffer[index],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 1]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 2]],
				intersection_point,
				current_t);

			if (!intersected_something || current_t < 0 || current_t >= current_ray->t) {
				continue;
			}

			current_ray->t = current_t;
			current_ray->intersected_material = scene.t_mats_gpu[i];
		}
#endif

		if (current_ray->intersected_material.type == 1) {
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);
		}
	}
}

//Launcher function
hipError_t launch_kernels(hipArray_const_t array, vec4* frame_buffer, KernelParams& kernel_params, BVH* bvh, int ray_buffer_size, int frame, bool new_frame) {

	hipError_t err = cudaAssert(BindSurfaceToArray(screen, array));
	if (err) {
		return err;
	}

	if (new_frame) {
		ResetAllVariables << <1, 1 >> > ();
	}else{
	}

	GeneratePrimaryRays << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.top_left, kernel_params.step_h, kernel_params.step_v, kernel_params.c_position, kernel_params.ray_buffer, ray_buffer_size, frame, frame_buffer);
	SetGlobalVariables << <1, 1 >> > (ray_buffer_size);
	Extend << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, bvh->cf_mbvh_gpu, bvh->mbvh_triangle_indices_gpu, kernel_params.ray_buffer, ray_buffer_size, kernel_params.scene.tri_count, frame);
#ifdef USE_REFERENCE	
	ShadeReference << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, kernel_params.ray_buffer_next, ray_buffer_size, frame_buffer);
#else
	Shade << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.shadow_ray_buffer, kernel_params.ray_buffer, kernel_params.ray_buffer_next, ray_buffer_size, kernel_params.scene.tri_count, frame, frame_buffer);
	Connect << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, bvh->cf_mbvh_gpu, bvh->mbvh_triangle_indices_gpu, kernel_params.shadow_ray_buffer, kernel_params.scene.tri_count, frame_buffer);
#endif
	//cudaAssert(DeviceSynchronize());

	std::swap(kernel_params.ray_buffer, kernel_params.ray_buffer_next);
	dim3 threads = dim3(16, 16);
	dim3 blocks = dim3((SCRWIDTH + threads.x - 1) / threads.x, (SCRHEIGHT + threads.y - 1) / threads.y);
	draw_frame << <blocks, threads >> > (frame_buffer, kernel_params.scene);
	cudaAssert(DeviceSynchronize());

	hipError_t c = hipError_t();
	return c;
}