#include "hip/hip_runtime.h"
#pragma once
#include <precomp.h>
#include <surface_functions.h>
using namespace glm;

//kernel variables
__device__ int counter_primary = 0;
__device__ int counter_extend = 0;
__device__ int counter_shade = 0;
__device__ int counter_connect = 0;
__device__ int start_position = 0;
__device__ int count_shadow_ray = 0;
__device__ int connect_ray_index = 0;
__device__ int helper_count = 0;

__device__ int active_paths_gpu = 0;
//debug variables
__device__ int debug_count = 0;

//draw variables
surface<void, cudaSurfaceType2D> screen;


/**DEVICE FUNCTIONS**/

//get random int
__device__ unsigned int random_int(unsigned int& seed){
	seed ^= seed << 13;
	seed ^= seed << 17;
	seed ^= seed << 5;
	return seed;
}

//get random float
__device__ float random_float(unsigned int& seed){
	return float(random_int(seed)) * 2.3283064365387e-10f;
}

__device__ vec3 cosineWeightedSample(vec3 normal, unsigned int& seed) {
	float rand1 = random_float(seed);
	float rand2 = random_float(seed);

	const float r = sqrtf(rand1);
	const float theta = DOUBLEPI * rand2;

	const float x = r * cosf(theta);
	const float z = r * sinf(theta);

	vec3 sampleDirection = vec3(x, sqrtf(max(0.0f, 1 - rand1)), z);
	sampleDirection = normalize(sampleDirection);
	vec3 Nb;
	vec3 Nt;

	//createing coordinate systen around the normal
	if (fabs(normal.x) > fabs(normal.y)) {
		float denom = 1 / sqrtf(normal.x * normal.x + normal.z * normal.z);
		Nt = vec3(normal.z, 0, -1.0f * normal.x) * denom;
	}
	else {
		float denom = 1 / sqrtf(normal.y * normal.y + normal.z * normal.z);
		Nt = vec3(0, -1.0f * normal.z, normal.y) * denom;
	}
	Nb = cross(normal, Nt);

	Nt = vec3(normal.y * Nb.z - normal.z * Nb.y, normal.z * Nb.x - normal.x * Nb.z, normal.x * Nb.y - normal.y * Nb.x);

	vec3 result = vec3(sampleDirection.x * Nb.x + sampleDirection.y * normal.x + sampleDirection.z * Nt.x, sampleDirection.x * Nb.y + sampleDirection.y * normal.y + sampleDirection.z * Nt.y, sampleDirection.x * Nb.z + sampleDirection.y * normal.z + sampleDirection.z * Nt.z);
	result = normalize(result);
	return result;
}

//take uniform random sample from hemisphere
__device__ vec3 sampleHemisphere(vec3 normal, unsigned int& seed){
	float rand1 = random_float(seed);
	float rand2 = random_float(seed);

	float sinTheta = sqrtf(1 - rand1 * rand1);
	float phi = DOUBLEPI * rand2;
	float x = sinTheta * cosf(phi);
	float z = sinTheta * sinf(phi);
	vec3 sampleDirection = vec3(x, rand1, z);
	vec3 Nb;
	vec3 Nt;

	//creating coordinate systen around the normal
	if (fabs(normal.x) > fabs(normal.y)){
		float denom = 1 / sqrtf(normal.x * normal.x + normal.z * normal.z);
		Nt = vec3(normal.z, 0, -1.0f * normal.x) * denom;
	} else{
		float denom = 1 / sqrtf(normal.y * normal.y + normal.z * normal.z);
		Nt = vec3(0, -1.0f * normal.z, normal.y) * denom;
	}
	Nb = cross(normal, Nt);

	Nt = vec3(normal.y * Nb.z - normal.z * Nb.y, normal.z * Nb.x - normal.x * Nb.z, normal.x * Nb.y - normal.y * Nb.x);

	vec3 result = vec3(sampleDirection.x * Nb.x + sampleDirection.y * normal.x + sampleDirection.z * Nt.x, sampleDirection.x * Nb.y + sampleDirection.y * normal.y + sampleDirection.z * Nt.y, sampleDirection.x * Nb.z + sampleDirection.y * normal.z + sampleDirection.z * Nt.z);
	result = normalize(result);
	return result;
}

//triangle intersection
__device__ bool intersect_triangle(Ray ray, vec3 v0, vec3 v1, vec3 v2, vec3& intersection_point, float& t){
	const float epsilon = 0.0000001;
	vec3 edge1, edge2, h, s, q;
	float a, f, u, v;
	edge1 = v1 - v0;
	edge2 = v2 - v0;
	h = cross(ray.direction, edge2);
	a = dot(edge1, h);

	if (a > -epsilon && a < epsilon)
		return false;

	f = 1.0 / a;
	s = ray.origin - v0;
	u = f * (dot(s, h));
	if (u < 0.0 || u > 1.0)
		return false;

	q = cross(s, edge1);
	v = f * dot(ray.direction, q);
	if (v < 0.0 || u + v > 1.0)
		return false;

	t = f * dot(edge2, q);
	intersection_point = ray.origin + ray.direction * t;
	if (t > epsilon){
		return true;
	} else
		return false;
}

//draw to cuda surface
__device__ void Draw(vec4& colour, int x, int y){
	surf2Dwrite(colour, screen, x * sizeof(vec4), y);
}

/**DEBUG KERNELS/FUNCTIONS**/

//generic setup for printing things
__global__ void print_helper(Ray* ray_buffer){
	printf("%i %i \n", count_shadow_ray, connect_ray_index);
}

/**MAIN KERNELS**/

//reset kernel variables
__global__ void SetGlobalVariables(int ray_buffer_size){
	counter_primary = 0;
	counter_extend = 0;
	counter_shade = 0;
	counter_connect = 0;
	connect_ray_index = 0;
	debug_count = 0;
	count_shadow_ray = 0;
}

__global__ void SetupNextIteration(int* m_a_p){
	active_paths_gpu = connect_ray_index;
	helper_count = 0;
	*m_a_p = active_paths_gpu;
}

//process and draw each pixel colour
__global__ void draw_frame(vec4* frame_buffer){

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= SCRWIDTH || y >= SCRHEIGHT) return;

	const int index = x + (y * SCRWIDTH);
	vec3 temp_colour = vec3();

	atomicAdd(&(frame_buffer[index].a), 1.0f);

	//sample counter is stored in .a 
	temp_colour.r = frame_buffer[index].r / frame_buffer[index].a;
	temp_colour.g = frame_buffer[index].g / frame_buffer[index].a;
	temp_colour.b = frame_buffer[index].b / frame_buffer[index].a;

	vec3 exponent = vec3(1.0f / 2.2f);
	vec4 colour = vec4(pow(temp_colour, exponent), 1.0f);

	Draw(colour, y, x);
}

//genereate kernel
__global__ void GeneratePrimaryRays(Scene scene, vec3 topLeft, vec3 stepH, vec3 stepV, vec3 c_position, Ray* ray_buffer, int ray_buffer_size, int frame){

	while (true){
		int index = atomicAdd(&counter_primary, 1);
		int buffer_index = index + connect_ray_index;

		if (index >= ray_buffer_size){
			return;
		}

		const int x = (index) % SCRWIDTH;
		const int y = (index / SCRWIDTH) % SCRHEIGHT;

		vec3 pixelPoint = vec3(topLeft.x, topLeft.y, topLeft.z) + (stepH * (y + 0.5f)) + (stepV * (x + 0.5f));
		vec3 rayDirection = vec3(pixelPoint.x - c_position.x, pixelPoint.y - c_position.y, pixelPoint.z - c_position.z);
		vec3 rayOrigin = vec3(c_position.x, c_position.y, c_position.z);
		rayDirection = normalize(rayDirection);

		ray_buffer[index] = Ray(rayOrigin, rayDirection, x + (y * SCRWIDTH));
		atomicAdd(&active_paths_gpu, 1);
	}
}

//extend kernel
__global__ void Extend(Scene scene, Ray* ray_buffer, int ray_buffer_size, int triangle_count, int frame){
	while (true){
		int index = atomicAdd(&counter_extend, 1);
		unsigned int seed = (index + frame * 147565741) * 720898027 * index;
		if (index >= active_paths_gpu){
			return;
		}

		//no acceleration structure yet
		for (int i = 0; i < triangle_count; i++){
			float current_t;
			vec3 intersection_point;

			bool intersected_something = intersect_triangle(ray_buffer[index],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 1]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 2]],
				intersection_point,
				current_t);

			if (!intersected_something || current_t < 0 || current_t >= ray_buffer[index].t){
				continue;
			}

			ray_buffer[index].t = current_t;
			ray_buffer[index].intersection_point = intersection_point;

			vec3 normal = scene.t_normals_gpu[i];
			float dot_product = dot(ray_buffer[index].direction, normal);
			if (dot_product > 0.0f){
				normal *= -1.0f;
			}
			ray_buffer[index].reflected_direction = normalize(sampleHemisphere(normal, seed));
			ray_buffer[index].intersection_index = i;

			float max_distance = MAXDISTANCE;

			if (ray_buffer[index].t < max_distance && ray_buffer[index].bounce <= MAXBOUNCE){
				ray_buffer[index].intersected_material = scene.t_mats_gpu[ray_buffer[index].intersection_index];
			}
		}
	}
}

//shade kernel
__global__ void Shade(Scene scene, Ray* shadow_ray_buffer, Ray* ray_buffer, Ray* ray_buffer_next, int ray_buffer_size, int triangle_count, int frame, vec4* frame_buffer){
	while (true){
		int index = atomicAdd(&counter_shade, 1);
		unsigned int seed = (index + frame * 147565741) * 720898027 * index;

		if (index >= active_paths_gpu){
			return;
		}
		Ray* current_ray = &ray_buffer[index];

		switch (current_ray->intersected_material.type){
			//Background
		case 0:
			current_ray->cumulative_colour = vec3(0);
			if (current_ray->bounce == 0){
				current_ray->cumulative_colour = scene.bg_colour;
			}

			current_ray->terminate_flag = true;
			break;
			//Light
		case 1:
			current_ray->cumulative_colour = vec3(0);
			if(current_ray->bounce == 0){
				current_ray->cumulative_colour = vec3(1.0f, 1.0f, 1.0f);
			}
			current_ray->terminate_flag = true;
			break;
			//Labertian
		case 2:
			float rand = random_float(seed);
			float split = 0;

			int counter = 1;
			vec3 random_point = vec3(0.0f);
			while (counter <= scene.light_tri_count){
				split += scene.light_areas_gpu[counter - 1];
				float proportion = split / scene.total_light_area;

				if (proportion > rand){

					//get random point on the light
					vec3 va = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3)]];
					vec3 vb = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3) + 1]];
					vec3 vc = scene.t_vertices_gpu[scene.t_indices_gpu[scene.tri_count * 3 - (counter * 3) + 2]];
					vec3 ab = vb - va;
					vec3 ac = vc - va;

					float w1 = random_float(seed);
					float w2 = random_float(seed);

					random_point = va + (w1 * ab) + (w2 * ac);
					break;
				}
				counter++;
			}

			vec3 BRDF = vec3(current_ray->intersected_material.colour.r * INVPI, current_ray->intersected_material.colour.g * INVPI, current_ray->intersected_material.colour.b * INVPI);
			float inv_pdf_hemisphere_sample = DOUBLEPI;


			vec3 shadow_ray_direction = random_point - current_ray->intersection_point;
			float distance_sqared = dot(shadow_ray_direction, shadow_ray_direction);
			shadow_ray_direction = normalize(shadow_ray_direction);
			vec3 shadow_ray_origin = current_ray->intersection_point + (0.0001f * shadow_ray_direction);

			vec3 normal = scene.t_normals_gpu[current_ray->intersection_index];
			if (dot(current_ray->reflected_direction, normal) < 0.0f){
				normal *= -1.0f;
			}

			vec3 light_normal = scene.t_normals_gpu[scene.tri_count - counter];
			if (dot(-1.0f * shadow_ray_direction, light_normal) < 0.0f){
				light_normal *= -1.0f;
			}

			float n_dot_l = dot(normal, shadow_ray_direction);
			float ln_dot_l = dot(light_normal, -1.0f * shadow_ray_direction);
			if (ln_dot_l > 0 && n_dot_l > 0){
			
				float area = scene.light_areas_gpu[scene.light_tri_count - counter];
				float inverse_area_pdf = scene.total_light_area / area;
				float solid_angle = (area * (ln_dot_l)) / distance_sqared;

				float pdf1 = 1 / solid_angle;
				float pdf2 = 1 / inv_pdf_hemisphere_sample;

				float combined_pdf = ((pdf1 / (pdf1 + pdf2))*pdf1) + ((pdf2 / (pdf1 + pdf2)) * pdf2);


				vec3 shadow_colour = BRDF * scene.emission * inverse_area_pdf * solid_angle * n_dot_l * current_ray->cumulative_colour;
				int shadow_index = atomicAdd(&count_shadow_ray, 1);
				shadow_ray_buffer[shadow_index] = Ray(shadow_ray_origin, shadow_ray_direction, current_ray->pixel_index, shadow_colour);
				shadow_ray_buffer[shadow_index].isShadow = true;
			}
			vec3 addition = inv_pdf_hemisphere_sample * BRDF * (dot(current_ray->reflected_direction, normal));
			current_ray->cumulative_colour *= addition;
			break;
		default:
			break;
		}

		switch(current_ray->terminate_flag){
		case true:
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);
			break;
		case false:
			vec3 ray_origin = current_ray->intersection_point + (current_ray->reflected_direction * 0.00001f);
			int e_index = atomicAdd(&connect_ray_index, 1);
			ray_buffer_next[e_index] = Ray(ray_origin, current_ray->reflected_direction, current_ray->pixel_index, current_ray->cumulative_colour);
			ray_buffer_next[e_index].bounce = current_ray->bounce + 1;
			break;
		default:
			break;
		}
	}
}

__global__ void ShadeReference(Scene scene, Ray* ray_buffer, Ray* ray_buffer_next, int ray_buffer_size, vec4* frame_buffer){
	while (true){
		int index = atomicAdd(&counter_shade, 1);

		if (index >= active_paths_gpu){
			return;
		}
		Ray* current_ray = &ray_buffer[index];
		float max_distance = MAXDISTANCE;

		if (ray_buffer[index].t < max_distance && ray_buffer[index].bounce <= MAXBOUNCE){
			current_ray->intersected_material = scene.t_mats_gpu[current_ray->intersection_index];
		}

		switch (current_ray->intersected_material.type){
		//Background
		case 0:
			if (current_ray->bounce == 0){
				current_ray->cumulative_colour = scene.bg_colour;
			} else{
				current_ray->cumulative_colour = vec3(0.0f, 0.0f, 0.0f);
			}

			current_ray->terminate_flag = true;
			break;
		//Light
		case 1:
			current_ray->cumulative_colour = current_ray->cumulative_colour * scene.emission;
			current_ray->terminate_flag = true;
			break;
		//Labertian
		case 2:
			vec3 normal = scene.t_normals_gpu[current_ray->intersection_index];
			float dot_product = dot(current_ray->reflected_direction, normal);
			if (dot_product < 0.0f){
				normal *= -1.0f;
			}
			vec3 BRDF = vec3(current_ray->intersected_material.colour.r * INVPI, current_ray->intersected_material.colour.g * INVPI, current_ray->intersected_material.colour.b * INVPI);
			float inv_PDF = DOUBLEPI;

			vec3 addition = inv_PDF * BRDF * (dot(current_ray->reflected_direction, normal));
			current_ray->cumulative_colour *= addition;
			break;
		default:
			break;
		}

		switch (current_ray->terminate_flag){
		case true:
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);

			atomicAdd(&debug_count, 1);

			break;
		case false:
			vec3 ray_origin = current_ray->intersection_point + (current_ray->reflected_direction * 0.00001f);
			int e_index = atomicAdd(&connect_ray_index, 1);
			ray_buffer_next[e_index] = Ray(ray_origin, current_ray->reflected_direction, current_ray->pixel_index, current_ray->cumulative_colour);
			ray_buffer_next[e_index].bounce = current_ray->bounce + 1;
			break;
		default:
			break;
		}
	}
}

//connect kernel
__global__ void Connect(Scene scene, Ray* shadow_ray_buffer, int triangle_count, vec4* frame_buffer){
	while (true){
		int index = atomicAdd(&counter_connect, 1);

		if (index >= count_shadow_ray){
			return;
		}

		Ray* current_ray = &shadow_ray_buffer[index];
		for (int i = 0; i < triangle_count; i++){
			float current_t;
			vec3 intersection_point;

			bool intersected_something = intersect_triangle(shadow_ray_buffer[index],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 1]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 2]],
				intersection_point,
				current_t);
			
			if (!intersected_something || current_t < 0 || current_t >= current_ray->t){
				continue;
			}

			current_ray->t = current_t;
			current_ray->intersected_material = scene.t_mats_gpu[i];
		}

		if (current_ray->intersected_material.type == 1){
			atomicAdd(&(frame_buffer[current_ray->pixel_index].r), current_ray->cumulative_colour.r);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].g), current_ray->cumulative_colour.g);
			atomicAdd(&(frame_buffer[current_ray->pixel_index].b), current_ray->cumulative_colour.b);
		}
	}
}

__global__ void g_singleAnswer(int* answer){ *answer = 2; }

//Launcher function
hipError_t launch_kernels(hipArray_const_t array, vec4* frame_buffer,  KernelParams & kernel_params, int ray_buffer_size, int frame){
	
	hipError_t err = cudaAssert(BindSurfaceToArray(screen, array));
	if (err){
		return err;
	}



	int active_paths = BUFFERSIZE;
	cudaAssert(Memset(kernel_params.malloc_active_paths, 0, sizeof(int)));
	
	GeneratePrimaryRays << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.top_left, kernel_params.step_h, kernel_params.step_v, kernel_params.c_position, kernel_params.ray_buffer, ray_buffer_size, frame);
	
	while(active_paths > 0){
		SetGlobalVariables << <1, 1 >> > (ray_buffer_size);	
		Extend << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, ray_buffer_size, kernel_params.scene.tri_count, frame);
#if 1		
		Shade << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.shadow_ray_buffer, kernel_params.ray_buffer, kernel_params.ray_buffer_next, ray_buffer_size, kernel_params.scene.tri_count, frame, frame_buffer);
		//print_helper << <1, 1 >> > (kernel_params.ray_buffer);
		Connect << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.shadow_ray_buffer, kernel_params.scene.tri_count, frame_buffer);
#else
		ShadeReference << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, kernel_params.ray_buffer_next, ray_buffer_size, frame_buffer);
#endif
		SetupNextIteration << <1, 1 >> > (kernel_params.malloc_active_paths);
		hipMemcpy(&active_paths, kernel_params.malloc_active_paths, sizeof(int), hipMemcpyDeviceToHost);

		cudaAssert(DeviceSynchronize());
		std::swap(kernel_params.ray_buffer, kernel_params.ray_buffer_next);
	}

	dim3 threads = dim3(16, 16);
 	dim3 blocks = dim3((SCRWIDTH + threads.x - 1) / threads.x, (SCRHEIGHT + threads.y - 1) / threads.y);
	draw_frame << <blocks, threads >> > (frame_buffer);
	cudaAssert(DeviceSynchronize());

	hipError_t c = hipError_t();
	return c;
}