#include "hip/hip_runtime.h"
#pragma once
#include <precomp.h>
#include <surface_functions.h>
using namespace glm;

//kernel variables
__device__ int counter_primary = 0;
__device__ int counter_extend = 0;
__device__ int counter_shade = 0;
__device__ int counter_connect = 0;
__device__ int start_position = 0;
__device__ int connect_ray_index = 0;

__device__ int debug_count = 0;

//draw variables
surface<void, cudaSurfaceType2D> screen;


//Device functions
__device__ unsigned int random_int(unsigned int& seed){
	seed ^= seed << 13;
	seed ^= seed << 17;
	seed ^= seed << 5;
	return seed;
}

__device__ float random_float(unsigned int& seed){
	return float(random_int(seed)) * 2.3283064365387e-10f;
}

__device__ vec3 sampleHemisphere(vec3 normal, unsigned int& seed){
	float rand1 = random_float(seed);
	float rand2 = random_float(seed);

	float sinTheta = sqrtf(1 - rand1 * rand1);
	float phi = DOUBLEPI * rand2;
	float x = sinTheta * cosf(phi);
	float z = sinTheta * sinf(phi);
	vec3 sampleDirection = vec3(x, rand1, z);
	vec3 Nb;
	vec3 Nt;

	//creating coordinate systen around the normal
	if (fabs(normal.x) > fabs(normal.y)){
		float denom = 1 / sqrtf(normal.x * normal.x + normal.z * normal.z);
		Nt = vec3(normal.z, 0, -1.0f * normal.x) * denom;
	} else{
		float denom = 1 / sqrtf(normal.y * normal.y + normal.z * normal.z);
		Nt = vec3(0, -1.0f * normal.z, normal.y) * denom;
	}
	Nb = cross(normal, Nt);

	Nt = vec3(normal.y * Nb.z - normal.z * Nb.y, normal.z * Nb.x - normal.x * Nb.z, normal.x * Nb.y - normal.y * Nb.x);

	vec3 result = vec3(sampleDirection.x * Nb.x + sampleDirection.y * normal.x + sampleDirection.z * Nt.x, sampleDirection.x * Nb.y + sampleDirection.y * normal.y + sampleDirection.z * Nt.y, sampleDirection.x * Nb.z + sampleDirection.y * normal.z + sampleDirection.z * Nt.z);
	result = normalize(result);
	return result;
}

__device__ bool intersect_triangle(Ray ray, vec3 v0, vec3 v1, vec3 v2, vec3& intersection_point, float& t){
	const float epsilon = 0.0000001;
	vec3 edge1, edge2, h, s, q;
	float a, f, u, v;
	edge1 = v1 - v0;
	edge2 = v2 - v0;
	h = cross(ray.direction, edge2);
	a = dot(edge1, h);

	if (a > -epsilon && a < epsilon)
		return false;

	f = 1.0 / a;
	s = ray.origin - v0;
	u = f * (dot(s, h));
	if (u < 0.0 || u > 1.0)
		return false;

	q = cross(s, edge1);
	v = f * dot(ray.direction, q);
	if (v < 0.0 || u + v > 1.0)
		return false;

	t = f * dot(edge2, q);
	intersection_point = ray.origin + ray.direction * t;
	if (t > epsilon){
		return true;
	} else
		return false;
}

//Debug Kernel
__global__ void print_frame(vec4* framebf, Ray* ray_buffer_next, Scene scene, int frame){
	//if(frame >=2 ){
	//for (int j = 0; j < connect_ray_index; j++){
	//	for (int i = 0; i < 4; i++){
	//		float current_t;
	//		vec3 intersection_point;

	//		bool intersected_something = intersect_triangle(ray_buffer_next[j],
	//			scene.t_vertices_gpu[scene.t_indices_gpu[i * 3]],
	//			scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 1]],
	//			scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 2]],
	//			intersection_point,
	//			current_t);

	//		if (i > 1 && intersected_something){
	//			printf("Yes \n");
	//		} else{
	//			//printf("no \n");
	//		}

	//	}
	//}
	//}

}

//Main Kernels
__global__ void set_global_variables(int ray_buffer_size){
	const unsigned int last_frame_stop = ray_buffer_size - counter_primary;
	start_position += last_frame_stop;
	start_position = start_position % (SCRWIDTH * SCRHEIGHT);

	counter_primary = 0;
	counter_extend = 0;
	counter_shade = 0;
	counter_connect = 0;
	connect_ray_index = 0;
	debug_count = 0;
}

__device__ void draw(vec4& colour, int x, int y){
	surf2Dwrite(colour, screen, x * sizeof(vec4), y);
}

__global__ void draw_frame(vec4* frame_buffer){
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= SCRWIDTH || y >= SCRHEIGHT) return;

	const int index = x + (y * SCRWIDTH);
	vec3 temp_colour = vec3();
	temp_colour.r = frame_buffer[index].r / frame_buffer[index].a;
	temp_colour.g = frame_buffer[index].g / frame_buffer[index].a;
	temp_colour.b = frame_buffer[index].b / frame_buffer[index].a;

	vec4 colour = vec4(temp_colour, 1.0f);
	draw(colour, y, x);
}

__global__ void colour(vec4* frame_buffer, int frame){

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * SCRWIDTH);

	if (x >= SCRWIDTH || y >= SCRHEIGHT){
		return;
	}
	unsigned int seed = (index + frame * 147565741) * 720898027 * index;
	frame_buffer[index] = vec4(random_float(seed), random_float(seed), random_float(seed), 1.0f);
}

__global__ void generatePrimaryRays(Scene scene, vec3 topLeft, vec3 stepH, vec3 stepV, vec3 c_position, Ray* ray_buffer, int ray_buffer_size, int frame){

	while (true){
		int index = atomicAdd(&counter_primary, 1);
		int buffer_index = index + connect_ray_index;

		if (buffer_index >= ray_buffer_size){
			return;
		}

		const int x = (start_position + index) % SCRWIDTH;
		const int y = ((start_position + index) / SCRWIDTH) % SCRHEIGHT;

		vec3 pixelPoint = vec3(topLeft.x, topLeft.y, topLeft.z) + (stepH * (y + 0.5f)) + (stepV * (x + 0.5f));
		vec3 rayDirection = vec3(pixelPoint.x - c_position.x, pixelPoint.y - c_position.y, pixelPoint.z - c_position.z);
		vec3 rayOrigin = vec3(c_position.x, c_position.y, c_position.z);
		rayDirection = normalize(rayDirection);

		
		ray_buffer[buffer_index] = Ray(rayOrigin, rayDirection, x + (y * SCRWIDTH));

	}
}

__global__ void extend(Scene scene, Ray* ray_buffer, int ray_buffer_size, int triangle_count, int frame){
	while (true){
		int index = atomicAdd(&counter_extend, 1);
		unsigned int seed = (index + frame * 147565741) * 720898027 * index;
		if (index >= ray_buffer_size){
			return;
		}

		//no acceleration structure yet
		for (int i = 0; i < triangle_count; i++){
			float current_t;
			vec3 intersection_point;

			bool intersected_something = intersect_triangle(ray_buffer[index],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 1]],
				scene.t_vertices_gpu[scene.t_indices_gpu[i * 3 + 2]],
				intersection_point,
				current_t);

			if (!intersected_something || current_t < 0 || current_t >= ray_buffer[index].t){
				continue;
			}

			ray_buffer[index].t = current_t;
			ray_buffer[index].intersection_point = intersection_point;

			vec3 normal = scene.t_normals_gpu[i];
			float dot_product = dot(ray_buffer[index].direction, normal);
			if (dot_product > 0.0f){
				normal *= -1.0f;
			}
			ray_buffer[index].reflected_direction = normalize(sampleHemisphere(normal, seed));
			ray_buffer[index].intersection_index = i;
		}
	}
}

__global__ void shade(Scene scene, Ray* ray_buffer, int ray_buffer_size){
	while (true){
		int index = atomicAdd(&counter_shade, 1);

		if (index >= ray_buffer_size){
			return;
		}
		Ray* current_ray = &ray_buffer[index];
		float max_distance = MAXDISTANCE;

		if (ray_buffer[index].t < max_distance && ray_buffer[index].bounce <= MAXBOUNCE){
			current_ray->intersected_material = scene.t_mats_gpu[current_ray->intersection_index];
		}

		switch (current_ray->intersected_material.type){
			//Background
		case 0:
			current_ray->cumulative_colour = vec3(0.0f, 0.0f, 0.0f);
			current_ray->terminate_flag = true;
			break;
			//Light
		case 1:
			current_ray->cumulative_colour *= scene.emission;
			current_ray->terminate_flag = true;
			break;
			//Labertian
		case 2:
			vec3 normal = scene.t_normals_gpu[current_ray->intersection_index];
			float dot_product = dot(current_ray->direction, normal);
			if (dot_product > 0.0f){
				normal *= -1.0f;
			}
			vec3 BRDF = vec3(current_ray->intersected_material.colour.r * INVPI, current_ray->intersected_material.colour.g * INVPI, current_ray->intersected_material.colour.b * INVPI);
			vec3 inv_PDF = vec3(DOUBLEPI, DOUBLEPI, DOUBLEPI);
			current_ray->cumulative_colour = inv_PDF * BRDF * current_ray->cumulative_colour * (dot(current_ray->reflected_direction, normal));
			break;
		default:
			break;
		}
	}
}

__global__ void connect(Scene scene, Ray* ray_buffer, Ray* ray_buffer_next, int ray_buffer_size, int triangle_count, vec4* frame_buffer){
	while (true){
		int index = atomicAdd(&counter_connect, 1);

		if (index >= ray_buffer_size){
			return;
		}

		Ray* current_ray = &ray_buffer[index];
		switch (current_ray->terminate_flag){
		case true:
			//clamp to 255
			if (current_ray->cumulative_colour.r > 255.0f)
				current_ray->cumulative_colour.r = 255.0f;
			if (current_ray->cumulative_colour.g > 255.0f)
				current_ray->cumulative_colour.g = 255.0f;
			if (current_ray->cumulative_colour.b > 255.0f)
				current_ray->cumulative_colour.b = 255.0f;

			atomicAdd(&frame_buffer[current_ray->pixel_index].r, current_ray->cumulative_colour.r);
			atomicAdd(&frame_buffer[current_ray->pixel_index].g, current_ray->cumulative_colour.g);
			atomicAdd(&frame_buffer[current_ray->pixel_index].b, current_ray->cumulative_colour.b);
			atomicAdd(&frame_buffer[current_ray->pixel_index].a, 1.0f);
			atomicAdd(&debug_count, 1);
			break;
		case false:
			vec3 ray_origin = current_ray->intersection_point + (current_ray->reflected_direction * 0.00001f);
			int e_index = atomicAdd(&connect_ray_index, 1);
			ray_buffer_next[e_index] = Ray(ray_origin, current_ray->reflected_direction, current_ray->pixel_index, current_ray->cumulative_colour);
			ray_buffer_next[e_index].bounce = current_ray->bounce + 1;
			atomicAdd(&frame_buffer[current_ray->pixel_index].a, 1.0f);
			break;
		default:
			break;
		}
	}
}

hipError_t print(hipArray_const_t array, vec4* frame_buffer, KernelParams & kernel_params, int ray_buffer_size, int frame){
	print_frame << <1, 1 >> > (frame_buffer, kernel_params.ray_buffer, kernel_params.scene, frame);
	cudaAssert(DeviceSynchronize());
	hipError_t c = hipError_t();
	return c;
}

hipError_t launch_kernels(hipArray_const_t array, vec4* frame_buffer,  KernelParams & kernel_params, int ray_buffer_size, int frame){
	
	hipError_t err = cudaAssert(BindSurfaceToArray(screen, array));
	if (err){
		return err;
	}

	generatePrimaryRays << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.top_left, kernel_params.step_h, kernel_params.step_v, kernel_params.c_position, kernel_params.ray_buffer, ray_buffer_size, frame);
	set_global_variables << <1, 1 >> > (ray_buffer_size);
	extend << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, ray_buffer_size, kernel_params.scene.tri_count, frame);
	shade << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, ray_buffer_size);
	connect << <kernel_params.sm_cores * 8, 128 >> > (kernel_params.scene, kernel_params.ray_buffer, kernel_params.ray_buffer_next, ray_buffer_size, kernel_params.scene.tri_count, frame_buffer);
	cudaAssert(DeviceSynchronize());

	std::swap(kernel_params.ray_buffer, kernel_params.ray_buffer_next);

	//const dim3 blockSize2d(8, 8);
	//const dim3 blocksPerGrid2d(
	//	(SCRWIDTH + blockSize2d.x - 1) / blockSize2d.x,
	//	(SCRHEIGHT + blockSize2d.y - 1) / blockSize2d.y);
	//colour << <blocksPerGrid2d, blockSize2d >> > (frame_buffer, frame);

	dim3 threads = dim3(16, 16);
 	dim3 blocks = dim3((SCRWIDTH + threads.x - 1) / threads.x, (SCRHEIGHT + threads.y - 1) / threads.y);
	draw_frame << <blocks, threads >> > (frame_buffer);
	cudaAssert(DeviceSynchronize());

	hipError_t c = hipError_t();
	return c;
}